#include "hip/hip_runtime.h"
#include "Gravity_GPU.cuh"
#include <>//threadIdx.x
#include <cstdio>
#include <iostream>
#include<thrust/reduce.h>
#include <thrust/transform_reduce.h>   
//ceshi 
# include <thrust/copy.h>
#include <thrust/host_vector.h>
extern __constant__  double pi_gpu = 3.1415926; //Բ����
__device__ int  g_size_gpu;
__device__ int s_size_gpu;
__device__ double p1_gpu;
__constant__  double p2_gpu = 0;//ˮ��
__device__ int n_gpu;
__device__ int g_row_gpu;
__device__ int g_col_gpu;
__device__ double Dif_Lat_gpu;
__device__ double Dif_Lon_gpu;
__device__ int  skip_gpu;
__device__ double  x_gpu[34];
__device__ double  A_gpu[34];

//����doubleԭ�Ӻ���
#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600

#else
__device__ double atomicAdd(double* address, double val)
{
	unsigned long long int* address_as_ull =
		(unsigned long long int*)address;
	unsigned long long int old = *address_as_ull, assumed;

	do {
		assumed = old;
		old = atomicCAS(address_as_ull, assumed,
			__double_as_longlong(val +
				__longlong_as_double(assumed)));

		// Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
	} while (assumed != old);

	return __longlong_as_double(old);
}
#endif



void Gravity::SetGroundPoints(std::vector<PointXYZ> GroundPoints)
{
	g_size = GroundPoints.size();
	GroundPoints_ = (PointXYZ*)malloc(sizeof(PointXYZ) * g_size);
	for (int i = 0; i < GroundPoints.size(); i++)
	{
		GroundPoints_[i] = GroundPoints[i];
	}
}

void Gravity::SetSatellitePoints(std::vector<PointXYZ> SatellitePoints)
{
	s_size = SatellitePoints.size();
	SatellitePoints_ = (PointXYZ*)malloc(sizeof(PointXYZ) * s_size);
	for (int i = 0; i < s_size; i++)
	{
		SatellitePoints_[i] = SatellitePoints[i];
	}
}

//void Gravity::SetRowandCol(const unsigned row__, const unsigned col__)
//{
//	g_row_ = row__;
//	g_col_ = col__;
//}

void Gravity::SetDensity(const double p)
{
	p_1 = p;
}

void Gravity::SetSplit()
{
	//����һ��cuda���豸���Խṹ��
	hipDeviceProp_t prop;
	//��ȡ��1��gpu�豸��������Ϣ
	hipGetDeviceProperties(&prop, 0);
	//�ж��������Ƿ񳬹���device�����������߳���
	size_t total = g_size * s_size;//��������
	for (split_ = 1; split_ < g_size; split_++)
	{
		if ((prop.maxGridSize[0] > (total / split_)) && is_Integer((double)g_size / split_))
			break;
	}
}

bool Gravity::is_Integer(double input)
{
	return input == (int)input;
}

void Gravity::Setn(const unsigned n)
{
	n_ = n;
	/*�ڵ�ϵ��*/
	if (n == 1)
	{
		double zero_points[1] = { 0.0000000000 };
		double W[1] = { 2.0000000000 };
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 2)
	{
		double zero_points[2] = {
		-0.5773502692,
		0.5773502692
		};
		double W[2] = {
		1.0000000000,
		1.0000000000
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 3)
	{
		double zero_points[3] = {
	-0.7745966692,
	0.0000000000,
	0.7745966692
		};
		double W[3] = {
		0.5555555556,
		0.8888888889,
		0.5555555556
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 4)
	{
		double zero_points[4] = {
		-0.8611363116,
		-0.3399810436,
		0.3399810436,
		0.8611363116
		};
		double W[4] = {
		0.3478548451,
		0.6521451549,
		0.6521451549,
		0.3478548451
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 5)
	{
		double zero_points[5] = {
		-0.9061798459,
		-0.5384693101,
		0.0000000000,
		0.5384693101,
		0.9061798459
		};
		double W[5] = {
		0.2369268851,
		0.4786286705,
		0.5688888889,
		0.4786286705,
		0.2369268851
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 6)
	{
		double zero_points[6] = {
		-0.9324695142,
		-0.6612093865,
		-0.2386191861,
		0.2386191861,
		0.6612093865,
		0.9324695142
		};
		double W[6] = {
		0.1713244924,
		0.3607615730,
		0.4679139346,
		0.4679139346,
		0.3607615730,
		0.1713244924
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 7)
	{
		double zero_points[7] = {
		-0.9491079123,
		-0.7415311856,
		-0.4058451514,
		0.0000000000,
		0.4058451514,
		0.7415311856,
		0.9491079123
		};
		double W[7] = {
		0.1294849662,
		0.2797053915,
		0.3818300505,
		0.4179591837,
		0.3818300505,
		0.2797053915,
		0.1294849662
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 8)
	{
		double zero_points[8] = {
		-0.9602898565,
		-0.7966664774,
		-0.5255324099,
		-0.1834346425,
		0.1834346425,
		0.5255324099,
		0.7966664774,
		0.9602898565
		};
		double W[8] = {
		0.1012285363,
		0.2223810345,
		0.3137066459,
		0.3626837834,
		0.3626837834,
		0.3137066459,
		0.2223810345,
		0.1012285363
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 9)
	{
		double zero_points[9] = {
		 -0.9681602395,
		 -0.8360311073,
		 -0.6133714327,
		 -0.3242534234,
		 0.0000000000,
		 0.3242534234,
		 0.6133714327,
		 0.8360311073,
		 0.9681602395
		};
		double W[9] = {
		0.0812743884,
		0.1806481607,
		0.2606106964,
		0.3123470770,
		0.3302393550,
		0.3123470770,
		0.2606106964,
		0.1806481607,
		0.0812743884
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 10)
	{
		double zero_points[10] = {
		-0.9739065285,
		-0.8650633667,
		-0.6794095683,
		-0.4333953941,
		-0.1488743390,
		0.1488743390,
		0.4333953941,
		0.6794095683,
		0.8650633667,
		0.9739065285
		};
		double W[10] = {
		0.0666713443,
		0.1494513492,
		0.2190863625,
		0.2692667193,
		0.2955242247,
		0.2955242247,
		0.2692667193,
		0.2190863625,
		0.1494513492,
		0.0666713443
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 11)
	{
		double zero_points[11] = {
		-0.9782286581,
		-0.8870625998,
		-0.7301520056,
		-0.5190961292,
		-0.2695431560,
		0.0000000000,
		0.2695431560,
		0.5190961292,
		0.7301520056,
		0.8870625998,
		0.9782286581
		};
		double W[11] = {
		0.0556685671,
		0.1255803695,
		0.1862902109,
		0.2331937646,
		0.2628045445,
		0.2729250868,
		0.2628045445,
		0.2331937646,
		0.1862902109,
		0.1255803695,
		0.0556685671
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}
	}
	else if (n == 12)
	{
		double zero_points[12] = {
		-0.9815606342,
		-0.9041172564,
		-0.7699026742,
		-0.5873179543,
		-0.3678314990,
		-0.1252334085,
		0.1252334085,
		0.3678314990,
		0.5873179543,
		0.7699026742,
		0.9041172564,
		0.9815606342
		};
		double W[12] = {
			0.0471753364,
			0.1069393260,
			0.1600783285,
			0.2031674267,
			0.2334925365,
			0.2491470458,
			0.2491470458,
			0.2334925365,
			0.2031674267,
			0.1600783285,
			0.1069393260,
			0.0471753364
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 13)
	{
		double zero_points[13] = {
		-0.9841830547,
		-0.9175983992,
		-0.8015780907,
		-0.6423493394,
		-0.4484927510,
		-0.2304583160,
		0.0000000000,
		0.2304583160,
		0.4484927510,
		0.6423493394,
		0.8015780907,
		0.9175983992,
		0.9841830547
		};
		double W[13] = {
		0.0404840048,
		0.0921214998,
		0.1388735102,
		0.1781459808,
		0.2078160475,
		0.2262831803,
		0.2325515532,
		0.2262831803,
		0.2078160475,
		0.1781459808,
		0.1388735102,
		0.0921214998,
		0.0404840048
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 14)
	{
		double zero_points[14] = {
		-0.9862838087,
		-0.9284348837,
		-0.8272013151,
		-0.6872929048,
		-0.5152486364,
		-0.3191123689,
		-0.1080549487,
		0.1080549487,
		0.3191123689,
		0.5152486364,
		0.6872929048,
		0.8272013151,
		0.9284348837,
		0.9862838087
		};
		double W[14] = {
		0.0351194603,
		0.0801580872,
		0.1215185707,
		0.1572031672,
		0.1855383975,
		0.2051984637,
		0.2152638535,
		0.2152638535,
		0.2051984637,
		0.1855383975,
		0.1572031672,
		0.1215185707,
		0.0801580872,
		0.0351194603
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 15)
	{
		double zero_points[15] = {
		-0.9879925180,
		-0.9372733924,
		-0.8482065834,
		-0.7244177314,
		-0.5709721726,
		-0.3941513471,
		-0.2011940940,
		0.0000000000,
		0.2011940940,
		0.3941513471,
		0.5709721726,
		0.7244177314,
		0.8482065834,
		0.9372733924,
		0.9879925180
		};
		double W[15] = {
		 0.0307532420,
		 0.0703660475,
		 0.1071592205,
		 0.1395706779,
		 0.1662692058,
		 0.1861610000,
		 0.1984314853,
		 0.2025782419,
		 0.1984314853,
		 0.1861610000,
		 0.1662692058,
		 0.1395706779,
		 0.1071592205,
		 0.0703660475,
		 0.0307532420
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 16)
	{
		double zero_points[16] = {
		-0.9894009350,
		-0.9445750231,
		-0.8656312024,
		-0.7554044084,
		-0.6178762444,
		-0.4580167777,
		-0.2816035508,
		-0.0950125098,
		0.0950125098,
		0.2816035508,
		0.4580167777,
		0.6178762444,
		0.7554044084,
		0.8656312024,
		0.9445750231,
		0.9894009350
		};
		double W[16] = {
		0.0271524594,
		0.0622535239,
		0.0951585117,
		0.1246289713,
		0.1495959888,
		0.1691565194,
		0.1826034150,
		0.1894506105,
		0.1894506105,
		0.1826034150,
		0.1691565194,
		0.1495959888,
		0.1246289713,
		0.0951585117,
		0.0622535239,
		0.0271524594
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 17)
	{
		double zero_points[17] = {
		-0.9905754753,
		-0.9506755218,
		-0.8802391537,
		-0.7815140039,
		-0.6576711592,
		-0.5126905371,
		-0.3512317635,
		-0.1784841815,
		0.0000000000,
		0.1784841815,
		0.3512317635,
		0.5126905371,
		0.6576711592,
		0.7815140039,
		0.8802391537,
		0.9506755218,
		0.9905754753
		};
		double W[17] = {
		0.0241483029,
		0.0554595294,
		0.0850361483,
		0.1118838472,
		0.1351363685,
		0.1540457611,
		0.1680041022,
		0.1765627054,
		0.1794464704,
		0.1765627054,
		0.1680041022,
		0.1540457611,
		0.1351363685,
		0.1118838472,
		0.0850361483,
		0.0554595294,
		0.0241483029
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 18)
	{
		double zero_points[18] = {
		-0.9915651684,
		-0.9558239496,
		-0.8926024665,
		-0.8037049590,
		-0.6916870431,
		-0.5597708311,
		-0.4117511615,
		-0.2518862257,
		-0.0847750130,
		0.0847750130,
		0.2518862257,
		0.4117511615,
		0.5597708311,
		0.6916870431,
		0.8037049590,
		0.8926024665,
		0.9558239496,
		0.9915651684
		};
		double W[18] = {
		0.0216160135,
		0.0497145489,
		0.0764257303,
		0.1009420441,
		0.1225552067,
		0.1406429147,
		0.1546846751,
		0.1642764837,
		0.1691423830,
		0.1691423830,
		0.1642764837,
		0.1546846751,
		0.1406429147,
		0.1225552067,
		0.1009420441,
		0.0764257303,
		0.0497145489,
		0.0216160135
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 19)
	{
		double zero_points[19] = {
		-0.9924068438,
		-0.9602081521,
		-0.9031559036,
		-0.8227146565,
		-0.7209661773,
		-0.6005453047,
		-0.4645707414,
		-0.3165641000,
		-0.1603586456,
		0.0000000000,
		0.1603586456,
		0.3165641000,
		0.4645707414,
		0.6005453047,
		0.7209661773,
		0.8227146565,
		0.9031559036,
		0.9602081521,
		0.9924068438
		};
		double W[19] = {
		0.0194617882,
		0.0448142268,
		0.0690445427,
		0.0914900216,
		0.1115666455,
		0.1287539625,
		0.1426067022,
		0.1527660421,
		0.1589688434,
		0.1610544498,
		0.1589688434,
		0.1527660421,
		0.1426067022,
		0.1287539625,
		0.1115666455,
		0.0914900216,
		0.0690445427,
		0.0448142268,
		0.0194617882
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 20)
	{
		double zero_points[20] = {
	-0.9931285992,
	-0.9639719273,
	-0.9122344283,
	-0.8391169718,
	-0.7463319065,
	-0.6360536807,
	-0.5108670020,
	-0.3737060887,
	-0.2277858511,
	-0.0765265211,
	0.0765265211,
	0.2277858511,
	0.3737060887,
	0.5108670020,
	0.6360536807,
	0.7463319065,
	0.8391169718,
	0.9122344283,
	0.9639719273,
	0.9931285992
		};
		double W[20] = {
	0.0176140071,
	0.0406014298,
	0.0626720483,
	0.0832767416,
	0.1019301198,
	0.1181945320,
	0.1316886384,
	0.1420961093,
	0.1491729865,
	0.1527533871,
	0.1527533871,
	0.1491729865,
	0.1420961093,
	0.1316886384,
	0.1181945320,
	0.1019301198,
	0.0832767416,
	0.0626720483,
	0.0406014298,
	0.0176140071
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 21)
	{
		double zero_points[21] = {
		-0.9937521706,
		-0.9672268386,
		-0.9200993342,
		-0.8533633646,
		-0.7684399635,
		-0.6671388042,
		-0.5516188359,
		-0.4243421202,
		-0.2880213168,
		-0.1455618542,
		0.0000000000,
		0.1455618542,
		0.2880213168,
		0.4243421202,
		0.5516188359,
		0.6671388042,
		0.7684399635,
		0.8533633646,
		0.9200993342,
		0.9672268386,
		0.9937521706
		};
		double W[21] = {
		0.0160172283,
		0.0369537898,
		0.0571344254,
		0.0761001136,
		0.0934444235,
		0.1087972992,
		0.1218314161,
		0.1322689386,
		0.1398873948,
		0.1445244040,
		0.1460811336,
		0.1445244040,
		0.1398873948,
		0.1322689386,
		0.1218314161,
		0.1087972992,
		0.0934444235,
		0.0761001136,
		0.0571344254,
		0.0369537898,
		0.0160172283
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 22)
	{
		double zero_points[22] = {
	-0.9942945855,
	-0.9700604978,
	-0.9269567722,
	-0.8658125777,
	-0.7878168060,
	-0.6944872632,
	-0.5876404035,
	-0.4693558380,
	-0.3419358209,
	-0.2078604267,
	-0.0697392733,
	0.0697392733,
	0.2078604267,
	0.3419358209,
	0.4693558380,
	0.5876404035,
	0.6944872632,
	0.7878168060,
	0.8658125777,
	0.9269567722,
	0.9700604978,
	0.9942945855
		};
		double W[22] = {
		0.0146279953,
		0.0337749016,
		0.0522933352,
		0.0697964684,
		0.0859416062,
		0.1004141444,
		0.1129322961,
		0.1232523768,
		0.1311735048,
		0.1365414983,
		0.1392518729,
		0.1392518729,
		0.1365414983,
		0.1311735048,
		0.1232523768,
		0.1129322961,
		0.1004141444,
		0.0859416062,
		0.0697964684,
		0.0522933352,
		0.0337749016,
		0.0146279953
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 23)
	{
		double zero_points[23] = {
		-0.9947693350,
		-0.9725424712,
		-0.9329710868,
		-0.8767523583,
		-0.8048884016,
		-0.7186613631,
		-0.6196098758,
		-0.5095014778,
		-0.3903010380,
		-0.2641356810,
		-0.1332568243,
		0.0000000000,
		0.1332568243,
		0.2641356810,
		0.3903010380,
		0.5095014778,
		0.6196098758,
		0.7186613631,
		0.8048884016,
		0.8767523583,
		0.9329710868,
		0.9725424712,
		0.9947693350
		};
		double W[23] = {
		0.0134118595,
		0.0309880059,
		0.0480376717,
		0.0642324214,
		0.0792814118,
		0.0929157661,
		0.1048920915,
		0.1149966402,
		0.1230490843,
		0.1289057222,
		0.1324620394,
		0.1336545722,
		0.1324620394,
		0.1289057222,
		0.1230490843,
		0.1149966402,
		0.1048920915,
		0.0929157661,
		0.0792814118,
		0.0642324214,
		0.0480376717,
		0.0309880059,
		0.0134118595
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 24)
	{
		double zero_points[24] = {
		-0.9951872200,
		-0.9747285560,
		-0.9382745520,
		-0.8864155270,
		-0.8200019860,
		-0.7401241916,
		-0.6480936519,
		-0.5454214714,
		-0.4337935076,
		-0.3150426797,
		-0.1911188675,
		-0.0640568929,
		0.0640568929,
		0.1911188675,
		0.3150426797,
		0.4337935076,
		0.5454214714,
		0.6480936519,
		0.7401241916,
		0.8200019860,
		0.8864155270,
		0.9382745520,
		0.9747285560,
		0.9951872200
		};
		double W[24] = {
	0.0123412298,
	0.0285313886,
	0.0442774388,
	0.0592985849,
	0.0733464814,
	0.0861901615,
	0.0976186521,
	0.1074442701,
	0.1155056681,
	0.1216704729,
	0.1258374563,
	0.1279381953,
	0.1279381953,
	0.1258374563,
	0.1216704729,
	0.1155056681,
	0.1074442701,
	0.0976186521,
	0.0861901615,
	0.0733464814,
	0.0592985849,
	0.0442774388,
	0.0285313886,
	0.0123412298
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 25)
	{
		double zero_points[25] = {
	-0.9955569698,
	-0.9766639215,
	-0.9429745712,
	-0.8949919979,
	-0.8334426288,
	-0.7592592630,
	-0.6735663685,
	-0.5776629302,
	-0.4730027314,
	-0.3611723058,
	-0.2438668837,
	-0.1228646926,
	0.0000000000,
	0.1228646926,
	0.2438668837,
	0.3611723058,
	0.4730027314,
	0.5776629302,
	0.6735663685,
	0.7592592630,
	0.8334426288,
	0.8949919979,
	0.9429745712,
	0.9766639215,
	0.9955569698
		};
		double W[25] = {
		  0.0113937985,
		  0.0263549866,
		  0.0409391567,
		  0.0549046960,
		  0.0680383338,
		  0.0801407003,
		  0.0910282620,
		  0.1005359491,
		  0.1085196245,
		  0.1148582591,
		  0.1194557635,
		  0.1222424430,
		  0.1231760537,
		  0.1222424430,
		  0.1194557635,
		  0.1148582591,
		  0.1085196245,
		  0.1005359491,
		  0.0910282620,
		  0.0801407003,
		  0.0680383338,
		  0.0549046960,
		  0.0409391567,
		  0.0263549866,
		  0.0113937985
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 26)
	{
		double zero_points[26] = {
			 -0.9958857011,
			 -0.9783854460,
			 -0.9471590667,
			 -0.9026378620,
			 -0.8454459428,
			 -0.7763859488,
			 -0.6964272604,
			 -0.6066922930,
			 -0.5084407148,
			 -0.4030517551,
			 -0.2920048395,
			 -0.1768588204,
			 -0.0592300934,
			 0.0592300934,
			 0.1768588204,
			 0.2920048395,
			 0.4030517551,
			 0.5084407148,
			 0.6066922930,
			 0.6964272604,
			 0.7763859488,
			 0.8454459428,
			 0.9026378620,
			 0.9471590667,
			 0.9783854460,
			 0.9958857011
		};
		//Gauss Weights
		double W[26] = {
		0.0105513726,
		0.0244178511,
		0.0379623833,
		0.0509758253,
		0.0632740463,
		0.0746841498,
		0.0850458943,
		0.0942138004,
		0.1020591611,
		0.1084718405,
		0.1133618165,
		0.1166604435,
		0.1183214153,
		0.1183214153,
		0.1166604435,
		0.1133618165,
		0.1084718405,
		0.1020591611,
		0.0942138004,
		0.0850458943,
		0.0746841498,
		0.0632740463,
		0.0509758253,
		0.0379623833,
		0.0244178511,
		0.0105513726
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 27)
	{
		double zero_points[27] = {
		-0.9961792629,
		-0.9799234760,
		-0.9509005578,
		-0.9094823207,
		-0.8562079080,
		-0.7917716391,
		-0.7170134737,
		-0.6329079719,
		-0.5405515646,
		-0.4411482518,
		-0.3359939036,
		-0.2264593654,
		-0.1139725856,
		0.0000000000,
		0.1139725856,
		0.2264593654,
		0.3359939036,
		0.4411482518,
		0.5405515646,
		0.6329079719,
		0.7170134737,
		0.7917716391,
		0.8562079080,
		0.9094823207,
		0.9509005578,
		0.9799234760,
		0.9961792629
		};
		//Gauss Weights
		double W[27] = {
		0.0097989961,
		0.0226862316,
		0.0352970538,
		0.0474494125,
		0.0589835369,
		0.0697488238,
		0.0796048678,
		0.0884231585,
		0.0960887274,
		0.1025016378,
		0.1075782858,
		0.1112524884,
		0.1134763461,
		0.1142208674,
		0.1134763461,
		0.1112524884,
		0.1075782858,
		0.1025016378,
		0.0960887274,
		0.0884231585,
		0.0796048678,
		0.0697488238,
		0.0589835369,
		0.0474494125,
		0.0352970538,
		0.0226862316,
		0.0097989961
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 28)
	{
		double zero_points[28] = {
		-0.9964424976,
		-0.9813031654,
		-0.9542592806,
		-0.9156330264,
		-0.8658925226,
		-0.8056413709,
		-0.7356108780,
		-0.6566510940,
		-0.5697204718,
		-0.4758742250,
		-0.3762515161,
		-0.2720616276,
		-0.1645692821,
		-0.0550792899,
		0.0550792899,
		0.1645692821,
		0.2720616276,
		0.3762515161,
		0.4758742250,
		0.5697204718,
		0.6566510940,
		0.7356108780,
		0.8056413709,
		0.8658925226,
		0.9156330264,
		0.9542592806,
		0.9813031654,
		0.9964424976
		};
		//Gauss Weights
		double W[28] = {
		0.0091242826,
		0.0211321126,
		0.0329014278,
		0.0442729348,
		0.0551073457,
		0.0652729240,
		0.0746462142,
		0.0831134172,
		0.0905717444,
		0.0969306580,
		0.1021129676,
		0.1060557659,
		0.1087111923,
		0.1100470130,
		0.1100470130,
		0.1087111923,
		0.1060557659,
		0.1021129676,
		0.0969306580,
		0.0905717444,
		0.0831134172,
		0.0746462142,
		0.0652729240,
		0.0551073457,
		0.0442729348,
		0.0329014278,
		0.0211321126,
		0.0091242826
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 29)
	{
		double zero_points[29] = {
		-0.9966794423,
		-0.9825455053,
		-0.9572855958,
		-0.9211802330,
		-0.8746378049,
		-0.8181854876,
		-0.7524628517,
		-0.6782145376,
		-0.5962817971,
		-0.5075929551,
		-0.4131528882,
		-0.3140316379,
		-0.2113522862,
		-0.1062782301,
		0.0000000000,
		0.1062782301,
		0.2113522862,
		0.3140316379,
		0.4131528882,
		0.5075929551,
		0.5962817971,
		0.6782145376,
		0.7524628517,
		0.8181854876,
		0.8746378049,
		0.9211802330,
		0.9572855958,
		0.9825455053,
		0.9966794423
		};
		//Gauss Weights
		double W[29] = {
		0.0085169039,
		0.0197320851,
		0.0307404922,
		0.0414020625,
		0.0515948269,
		0.0612030907,
		0.0701179333,
		0.0782383271,
		0.0854722574,
		0.0917377571,
		0.0969638341,
		0.1010912738,
		0.1040733101,
		0.1058761551,
		0.1064793817,
		0.1058761551,
		0.1040733101,
		0.1010912738,
		0.0969638341,
		0.0917377571,
		0.0854722574,
		0.0782383271,
		0.0701179333,
		0.0612030907,
		0.0515948269,
		0.0414020625,
		0.0307404922,
		0.0197320851,
		0.0085169039
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 30)
	{
		double zero_points[30] = {
		-0.9968934841,
		-0.9836681233,
		-0.9600218650,
		-0.9262000474,
		-0.8825605358,
		-0.8295657624,
		-0.7677774321,
		-0.6978504948,
		-0.6205261830,
		-0.5366241481,
		-0.4470337695,
		-0.3527047255,
		-0.2546369262,
		-0.1538699136,
		-0.0514718426,
		0.0514718426,
		0.1538699136,
		0.2546369262,
		0.3527047255,
		0.4470337695,
		0.5366241481,
		0.6205261830,
		0.6978504948,
		0.7677774321,
		0.8295657624,
		0.8825605358,
		0.9262000474,
		0.9600218650,
		0.9836681233,
		0.9968934841
		};
		//Gauss Weights
		double W[30] = {
		0.0079681925,
		0.0184664683,
		0.0287847079,
		0.0387991926,
		0.0484026728,
		0.0574931562,
		0.0659742299,
		0.0737559747,
		0.0807558952,
		0.0868997872,
		0.0921225222,
		0.0963687372,
		0.0995934206,
		0.1017623897,
		0.1028526529,
		0.1028526529,
		0.1017623897,
		0.0995934206,
		0.0963687372,
		0.0921225222,
		0.0868997872,
		0.0807558952,
		0.0737559747,
		0.0659742299,
		0.0574931562,
		0.0484026728,
		0.0387991926,
		0.0287847079,
		0.0184664683,
		0.0079681925
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 31)
	{
		//Gauss Nodes in [-1.0, 1.0]
		double zero_points[31] = {
		-0.9970874818,
		-0.9846859097,
		-0.9625039251,
		-0.9307569979,
		-0.8897600299,
		-0.8399203201,
		-0.7817331484,
		-0.7157767846,
		-0.6427067229,
		-0.5632491614,
		-0.4781937820,
		-0.3883859016,
		-0.2947180700,
		-0.1981211993,
		-0.0995553122,
		0.0000000000,
		0.0995553122,
		0.1981211993,
		0.2947180700,
		0.3883859016,
		0.4781937820,
		0.5632491614,
		0.6427067229,
		0.7157767846,
		0.7817331484,
		0.8399203201,
		0.8897600299,
		0.9307569979,
		0.9625039251,
		0.9846859097,
		0.9970874818
		};
		//Gauss Weights
		double W[31] = {
		0.0074708316,
		0.0173186208,
		0.0270090192,
		0.0364322739,
		0.0454937075,
		0.0541030824,
		0.0621747866,
		0.0696285832,
		0.0763903866,
		0.0823929918,
		0.0875767406,
		0.0918901139,
		0.0952902429,
		0.0977433354,
		0.0992250112,
		0.0997205448,
		0.0992250112,
		0.0977433354,
		0.0952902429,
		0.0918901139,
		0.0875767406,
		0.0823929918,
		0.0763903866,
		0.0696285832,
		0.0621747866,
		0.0541030824,
		0.0454937075,
		0.0364322739,
		0.0270090192,
		0.0173186208,
		0.0074708316
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 32)
	{
		//Gauss Nodes in [-1.0, 1.0]
		double zero_points[32] = { -0.9972638618,
			-0.9856115115,
			-0.9647622556,
			-0.9349060759,
			-0.8963211558,
			-0.8493676137,
			-0.7944837960,
			-0.7321821187,
			-0.6630442669,
			-0.5877157572,
			-0.5068999089,
			-0.4213512761,
			-0.3318686023,
			-0.2392873623,
			-0.1444719616,
			-0.0483076657,
			0.0483076657,
			0.1444719616,
			0.2392873623,
			0.3318686023,
			0.4213512761,
			0.5068999089,
			0.5877157572,
			0.6630442669,
			0.7321821187,
			0.7944837960,
			0.8493676137,
			0.8963211558,
			0.9349060759,
			0.9647622556,
			0.9856115115,
			0.9972638618
		};
		//Gauss Weights
		double W[32] = {
		0.0070186100,
		0.0162743947,
		0.0253920653,
		0.0342738629,
		0.0428358980,
		0.0509980593,
		0.0586840935,
		0.0658222228,
		0.0723457941,
		0.0781938958,
		0.0833119242,
		0.0876520930,
		0.0911738787,
		0.0938443991,
		0.0956387201,
		0.0965400885,
		0.0965400885,
		0.0956387201,
		0.0938443991,
		0.0911738787,
		0.0876520930,
		0.0833119242,
		0.0781938958,
		0.0723457941,
		0.0658222228,
		0.0586840935,
		0.0509980593,
		0.0428358980,
		0.0342738629,
		0.0253920653,
		0.0162743947,
		0.0070186100
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 33)
	{
		//Gauss Nodes in [-1.0, 1.0]
		double zero_points[33] = {
	  -0.9974246942,
	  -0.9864557262,
	  -0.9668229097,
	  -0.9386943726,
	  -0.9023167677,
	  -0.8580096527,
	  -0.8061623563,
	  -0.7472304964,
	  -0.6817319600,
	  -0.6102423458,
	  -0.5333899048,
	  -0.4518500173,
	  -0.3663392577,
	  -0.2776090972,
	  -0.1864392988,
	  -0.0936310659,
	  0.0000000000,
	  0.0936310659,
	  0.1864392988,
	  0.2776090972,
	  0.3663392577,
	  0.4518500173,
	  0.5333899048,
	  0.6102423458,
	  0.6817319600,
	  0.7472304964,
	  0.8061623563,
	  0.8580096527,
	  0.9023167677,
	  0.9386943726,
	  0.9668229097,
	  0.9864557262,
	  0.9974246942
		};
		//Gauss Weights
		double W[33] = {
		0.0066062278,
		0.0153217015,
		0.0239155481,
		0.0323003586,
		0.0404015413,
		0.0481477428,
		0.0554708466,
		0.0623064825,
		0.0685945728,
		0.0742798548,
		0.0793123648,
		0.0836478761,
		0.0872482876,
		0.0900819587,
		0.0921239866,
		0.0933564261,
		0.0937684462,
		0.0933564261,
		0.0921239866,
		0.0900819587,
		0.0872482876,
		0.0836478761,
		0.0793123648,
		0.0742798548,
		0.0685945728,
		0.0623064825,
		0.0554708466,
		0.0481477428,
		0.0404015413,
		0.0323003586,
		0.0239155481,
		0.0153217015,
		0.0066062278
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
	else if (n == 34)
	{
		//Gauss Nodes in [-1.0, 1.0]
		double zero_points[34] = {
		  -0.9975717538,
		  -0.9872278164,
		  -0.9687082625,
		  -0.9421623974,
		  -0.9078096777,
		  -0.8659346383,
		  -0.8168842279,
		  -0.7610648766,
		  -0.6989391132,
		  -0.6310217271,
		  -0.5578755007,
		  -0.4801065452,
		  -0.3983592778,
		  -0.3133110813,
		  -0.2256666916,
		  -0.1361523573,
		  -0.0455098220,
		  0.0455098220,
		  0.1361523573,
		  0.2256666916,
		  0.3133110813,
		  0.3983592778,
		  0.4801065452,
		  0.5578755007,
		  0.6310217271,
		  0.6989391132,
		  0.7610648766,
		  0.8168842279,
		  0.8659346383,
		  0.9078096777,
		  0.9421623974,
		  0.9687082625,
		  0.9872278164,
		  0.9975717538
		};
		//Gauss Weights
		double W[34] = {
		0.0062291406,
		0.0144501627,
		0.0225637220,
		0.0304913806,
		0.0381665938,
		0.0455256115,
		0.0525074146,
		0.0590541358,
		0.0651115216,
		0.0706293758,
		0.0755619747,
		0.0798684443,
		0.0835130997,
		0.0864657397,
		0.0887018978,
		0.0902030444,
		0.0909567403,
		0.0909567403,
		0.0902030444,
		0.0887018978,
		0.0864657397,
		0.0835130997,
		0.0798684443,
		0.0755619747,
		0.0706293758,
		0.0651115216,
		0.0590541358,
		0.0525074146,
		0.0455256115,
		0.0381665938,
		0.0304913806,
		0.0225637220,
		0.0144501627,
		0.0062291406
		};
		for (int i = 0; i < n; i++)
		{
			x[i] = zero_points[i];
			A[i] = W[i];
		}

	}
}

void Gravity::SetDif_Lat_and_Dif_Lon1(const double Dif_Lat, const double Dif_Lon)
{
	Dif_Lat_ = Dif_Lat;
	Dif_Lon_ = Dif_Lon;
}

void Gravity::CalculatGravitation(std::vector<double>& sum, enum function func)
{
	SetSplit();
	/*����*/
	int splited_s_size = s_size / split_;//��������ֿ�
	//int skip = 1;
	hipMemcpyToSymbol(HIP_SYMBOL(s_size_gpu), &splited_s_size, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(g_size_gpu), &g_size, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(p1_gpu), &p_1, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(Dif_Lat_gpu), &Dif_Lat_, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(Dif_Lon_gpu), &Dif_Lon_, sizeof(double));
	hipMemcpyToSymbol(HIP_SYMBOL(n_gpu), &n_, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(x_gpu), x, sizeof(double) * n_);
	hipMemcpyToSymbol(HIP_SYMBOL(A_gpu), A, sizeof(double) * n_);

	/*�����������*/
	int minGridSize;// ��������������С����
	int blockSize;
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, VZZ_GPU, 0, g_size * splited_s_size);
	int gridSize = (splited_s_size * g_size + blockSize - 1) / blockSize;//  

	/**************************************��������ֿ�*****************************/
	//����
	PointXYZ* GroundPoints_GPU;
	PointXYZ* SatellitePoints_GPU;
	PointXYZ* Temp_SatellitePoints_;
	double* Results;
	double* Results_GPU;
	//�����ڴ�
	Results = (double*)malloc(splited_s_size * sizeof(double));
	Temp_SatellitePoints_ = (PointXYZ*)malloc(splited_s_size * sizeof(PointXYZ));
	hipMalloc((void**)&GroundPoints_GPU, g_size * sizeof(PointXYZ));
	hipMalloc((void**)&SatellitePoints_GPU, splited_s_size * sizeof(PointXYZ));
	hipMalloc((void**)&Results_GPU, splited_s_size * sizeof(double));
	hipMemcpy(GroundPoints_GPU, GroundPoints_, g_size * sizeof(PointXYZ), hipMemcpyHostToDevice);

	for (size_t i = 0; i < split_; i++)
	{
		//�ֿ���������
		for (size_t j = 0; j < splited_s_size; j++)
		{
			Temp_SatellitePoints_[j] = SatellitePoints_[j + i * splited_s_size];
		}
		hipMemcpy(SatellitePoints_GPU, Temp_SatellitePoints_, splited_s_size * sizeof(PointXYZ), hipMemcpyHostToDevice);
		hipMemset(Results_GPU, 0, splited_s_size * sizeof(double));

		/*ѡ���ݶ�*/
		switch (func)
		{
		case VX:
			VXX_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		case VY:
			VXY_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		case VZ:
			VXZ_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		case VXX:
			VXX_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		case VXY:
			VXY_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		case VXZ:
			VXZ_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		case VYY:
			VYY_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		case VYZ:
			VYZ_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		case VZZ:
			VZZ_GPU << <gridSize, blockSize, 0 >> > (GroundPoints_GPU, SatellitePoints_GPU, Results_GPU);
			break;
		default:
			break;
		}
		hipMemcpy(Results, Results_GPU, sizeof(double) * splited_s_size, hipMemcpyDeviceToHost);
		for (int i = 0; i < splited_s_size; i++)
		{
			sum.push_back(Results[i]);
		}

	}

	/*�ͷ��Դ�*/
	hipFree(GroundPoints_);
	hipFree(GroundPoints_GPU);
	hipFree(SatellitePoints_GPU);
	hipFree(Temp_SatellitePoints_);
	hipFree(Results);
	hipFree(Results_GPU);
}

__global__ void VX_GPU(PointXYZ* GroundPoints, PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = tid + bid * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)//��450�ֽ�Ϊ45��10  / skip_gpu
	{
		int step = idx % s_size_gpu;//����ֱ�����
		int s = idx / s_size_gpu;

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;


		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vx_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);
				}
			}
		}
	}
}

__global__ void VY_GPU(PointXYZ* GroundPoints, PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = tid + bid * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)//��450�ֽ�Ϊ45��10  / skip_gpu
	{
		int step = idx % s_size_gpu;//����ֱ�����
		int s = idx / s_size_gpu;

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;


		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vy_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);
				}
			}
		}
	}
}

__global__ void VZ_GPU(PointXYZ* GroundPoints, PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = tid + bid * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)//��450�ֽ�Ϊ45��10  / skip_gpu
	{
		int step = idx % s_size_gpu;//����ֱ�����
		int s = idx / s_size_gpu;

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;


		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vz_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);
				}
			}
		}
	}
}

__global__  void VXX_GPU(PointXYZ* GroundPoints,
	PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = tid + bid * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)//��450�ֽ�Ϊ45��10  / skip_gpu
	{
		int step = idx % s_size_gpu;//����ֱ�����
		int s = idx / s_size_gpu;

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;


		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vxx_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);
				}
			}
		}
	}
}

__global__  void VXY_GPU(PointXYZ* GroundPoints,
	PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = tid + bid * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)
	{
		int step = idx % s_size_gpu;//����ֱ�����

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;

		int s = idx / s_size_gpu;
		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vxy_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);
				}
			}
		}
	}
}

__global__  void VXZ_GPU(PointXYZ* GroundPoints,
	PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = tid + bid * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)//��450�ֽ�Ϊ45��10  / skip_gpu
	{
		int step = idx % s_size_gpu;//����ֱ�����

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;

		//ÿ���ƶ�skip_gpu����
		//for (int s = skip_gpu * (idx / s_size_gpu); s < skip_gpu + skip_gpu * (idx / s_size_gpu); s++)
		int s = idx / s_size_gpu;
		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vxz_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);
				}
			}
		}
	}
}

__global__  void VYY_GPU(PointXYZ* GroundPoints,
	PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = tid + bid * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)//��450�ֽ�Ϊ45��10  / skip_gpu
	{
		int step = idx % s_size_gpu;//����ֱ�����

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;

		//ÿ���ƶ�skip_gpu����
		//for (int s = skip_gpu * (idx / s_size_gpu); s < skip_gpu + skip_gpu * (idx / s_size_gpu); s++)
		int s = idx / s_size_gpu;
		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vyy_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);
				}
			}
		}
	}
}

__global__  void VYZ_GPU(PointXYZ* GroundPoints,
	PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = tid + bid * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)//��450�ֽ�Ϊ45��10  / skip_gpu
	{
		int step = idx % s_size_gpu;//����ֱ�����

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;

		//ÿ���ƶ�skip_gpu����
		//for (int s = skip_gpu * (idx / s_size_gpu); s < skip_gpu + skip_gpu * (idx / s_size_gpu); s++)
		int s = idx / s_size_gpu;
		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vyz_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);
				}
			}
		}
	}
}

__global__  void VZZ_GPU(PointXYZ* GroundPoints,
	PointXYZ* SatellitePoints, double* result_)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < s_size_gpu * g_size_gpu)//��450�ֽ�Ϊ45��10  / skip_gpu
	{
		int step = idx % s_size_gpu;//����ֱ�����

		/*�Ƕ�ת�����Ƕ�ת���ȣ��߶�+����뾶*/
		double transfer = pi_gpu / 180;
		double Satellite_x_ = transfer * SatellitePoints[step].x;
		double Satellite_y_ = transfer * SatellitePoints[step].y;
		double Satellite_z_ = 6371000 + SatellitePoints[step].z;

		//ÿ���ƶ�skip_gpu����
		//for (int s = skip_gpu * (idx / s_size_gpu); s < skip_gpu + skip_gpu * (idx / s_size_gpu); s++)
		int s = idx / s_size_gpu;
		/*������������*�ܶ�*/
		double p_gpu;
		if (GroundPoints[s].z >= 0)
			p_gpu = p1_gpu;
		else
			p_gpu = p2_gpu;
		double const_Gp = 66.7 * p_gpu;

		/*���ػ���*/
		for (int i = 0; i < n_gpu; i++)
		{
			for (int j = 0; j < n_gpu; j++)
			{
				for (int k = 0; k < n_gpu; k++)
				{
					///*�����,ͨ���任����׼����[-1, 1]*/
					double suface_z = (GroundPoints[s].z * x_gpu[i] + GroundPoints[s].z) / 2 + 6371000;
					double suface_y = (Dif_Lon_gpu * x_gpu[j] + 2 * GroundPoints[s].y) * transfer / 2;
					double suface_x = (Dif_Lat_gpu * x_gpu[k] + 2 * GroundPoints[s].x) * transfer / 2;

					/*�������*/
					double temp = const_Gp * GroundPoints[s].z * Dif_Lat_gpu * transfer * Dif_Lon_gpu * transfer *
						A_gpu[i] * A_gpu[j] * A_gpu[k] * vzz_GPU(suface_x, suface_y, suface_z, Satellite_x_, Satellite_y_,
							Satellite_z_) / 8;//����һ��������ػ���
					atomicAdd(&result_[step], temp);

				}
			}
		}

	}
}

__device__ double vx_GPU(double suface_x, double suface_y, double suface_z, double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double k_fai = cos(Satellite_y) * sin(suface_y) - sin(Satellite_y) * cos(suface_y) * cos(suface_x - Satellite_x);
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = suface_z * k_fai * K / powf(L, 3);
	return f;
}

__device__ double vy_GPU(double suface_x, double suface_y, double suface_z, double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double k_fai = cos(Satellite_y) * sin(suface_y) - sin(Satellite_y) * cos(suface_y) * cos(suface_x - Satellite_x);
	double k_lamba = cos(Satellite_y) * cos(suface_y) * sin(suface_x - Satellite_x);
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = suface_z * k_lamba * K / (cos(Satellite_y) * powf(L, 3));
	return f;
}

__device__ double vz_GPU(double suface_x, double suface_y, double suface_z, double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double k_fai = cos(Satellite_y) * sin(suface_y) - sin(Satellite_y) * cos(suface_y) * cos(suface_x - Satellite_x);
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = (suface_z * cos_angle - Satellite_z) * K / powf(L, 3);
	return f;
}

__device__ double vxx_GPU(double suface_x, double suface_y, double suface_z,
	double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double k_fai = cos(Satellite_y) * sin(suface_y) - sin(Satellite_y) * cos(suface_y) * cos(suface_x - Satellite_x);
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = (3 * (suface_z * suface_z) * (k_fai * k_fai) / powf(L, 5) - 1 / powf(L, 3)) * K;
	return f;
}

__device__ double vxy_GPU(double suface_x, double suface_y, double suface_z,
	double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double k_fai = cos(Satellite_y) * sin(suface_y) - sin(Satellite_y) * cos(suface_y) * cos(suface_x - Satellite_x);
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = (3 * (suface_z * suface_z) * (k_fai * cos(suface_y) * sin(suface_x - Satellite_x)) / powf(L, 5)) * K;
	return f;
}

__device__ double vxz_GPU(double suface_x, double suface_y, double suface_z,
	double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double k_fai = cos(Satellite_y) * sin(suface_y) - sin(Satellite_y) * cos(suface_y) * cos(suface_x - Satellite_x);
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = (3 * suface_z * k_fai * (suface_z * cos_angle - Satellite_z) / powf(L, 5)) * K;
	return f;
}
__device__ double vyy_GPU(double suface_x, double suface_y, double suface_z,
	double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double k_fai = cos(Satellite_y) * sin(suface_y) - sin(Satellite_y) * cos(suface_y) * cos(suface_x - Satellite_x);
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = ((3 * suface_z * cos(suface_y) * sin(suface_x - Satellite_x) * suface_z * cos(suface_y) * sin(suface_x - Satellite_x) / powf(L, 5)) - 1 / powf(L, 3)) * K;
	return f;
}
__device__ double vyz_GPU(double suface_x, double suface_y, double suface_z,
	double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double k_fai = cos(Satellite_y) * sin(suface_y) - sin(Satellite_y) * cos(suface_y) * cos(suface_x - Satellite_x);
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = (3 * suface_z * cos(suface_y) * sin(suface_x - Satellite_x) * (suface_z * cos_angle - Satellite_z) / powf(L, 5)) * K;
	return f;
}

__device__ double vzz_GPU(double suface_x, double suface_y, double suface_z,
	double Satellite_x, double Satellite_y, double Satellite_z)
{
	double f;
	double cos_angle = sin(suface_y) * sin(Satellite_y) + cos(suface_y) * cos(Satellite_y) * cos(Satellite_x - suface_x);
	double L = sqrt(suface_z * suface_z + Satellite_z * Satellite_z - 2 * suface_z * Satellite_z * cos_angle);
	double K = suface_z * suface_z * cos(suface_y);
	f = (3 * (suface_z * cos_angle - Satellite_z) * (suface_z * cos_angle - Satellite_z) / powf(L, 5) - 1 / powf(L, 3)) * K;
	return f;
}

void read_satellites(const std::string file_name, std::vector<PointXYZ>& data)
{
	/*��ȡ�ļ�*/
	std::ifstream file(file_name);
	if (file.bad()) {
		std::cout << "���ļ�ʧ�ܣ�" << std::endl;
		return;
	}
	PointXYZ temp;
	while (!file.eof())
	{
		file >> temp.x >> temp.y >> temp.z;
		data.push_back(temp);
	}
	file.close();

	std::cout << "��ȡ�۲�����!" << std::endl;

}

void save_arc(const std::string save_file_name, std::vector<double>& I)
{
	std::ofstream outFile;
	//���ļ�
	outFile.open(save_file_name, std::ios::out);
	int sum = 0;
	for (int i = 0; i < I.size(); i++)
	{
		outFile << I[i];//д������
		outFile << "\n";//д������
	}
	//�ر��ļ�
	outFile.close();
}

void read_arc(const std::string file_name, std::vector<PointXYZ>& data, double& lat, double& lon)
{
	/*����*/
	PointXYZ p;
	int i = 0;
	unsigned skip_y = 1;
	char szBuf[20];
	double step;
	double temp_x;
	unsigned row_, col_;
	int stop = 0;
	/*��ȡ�ļ�*/
	std::ifstream file(file_name);
	if (file.bad()) {
		std::cout << "���ļ�ʧ�ܣ�" << std::endl;
		return;
	}

	/*��ȡ�������¼*/
	while (!file.eof())
	{
		if (i == 0)
		{
			file >> szBuf >> col_;
			i++;
			continue;
		}
		else if (i == 1)
		{
			file >> szBuf >> row_;
			i++;
			continue;

		}
		else if (i == 2)
		{
			file >> szBuf >> temp_x;
			p.x = temp_x;
			i++;
			continue;
		}
		else if (i == 3)
		{
			file >> szBuf >> p.y;
			i++;
			continue;
		}
		else if (i == 4)
		{
			file >> szBuf >> step;
			lat = step;/*����ά�Ⱦ���*/
			lon = step;/*����ά�Ⱦ���*/
			i++;
			continue;
		}
		else if (i == 5)
		{
			double skip;
			file >> szBuf >> skip;
			i++;
			continue;
		}
		else
		{
			file >> p.z;
			data.push_back(p);
			if (skip_y == (row_ * col_) || file.peek() == EOF)
			{
				break;
			}
			p.x += step;
			if (skip_y % col_ == 0)
			{
				stop++;
				p.y += step;
				p.x = temp_x;
			}
			skip_y++;
		}
	}
	file.close();

	std::cout << "��ȡ���ε����!" << std::endl;
}

